#include "hip/hip_runtime.h"
// Funções atômicas ajudam a resolver o problema de ter muitas threads acessando a mesma área de memória
// Operações atômicas garantem que somente uma thread esteja acessando uma área de memória em um dado momento
// Operações atômicas devem ser configuradas com sm_20_atomic_functions.h ou o padrão correspondente da arquitetura


#include "hip/hip_runtime.h"
#include ""
#include "sm_20_atomic_functions.h"

#include <iostream>
using namespace std;

__device__ int dSum = 0;

__global__ void sum(int* d)
{
	int tid = threadIdx.x;

	// Essa instrução vai gerar um problema, pois estamos tratando as threads como sequenciais, problema conhecido como race condition
	//dSum += d[tid];

	// A função atomicAdd evita o problema de race condition
	atomicAdd(&dSum, d[tid]);
}

int main()
{
	const int count = 256;
	const int size = count * sizeof(int);

	int h[count];

	for (int i = 0; i < count; ++i)
		h[i] = i + 1;

	int* d;

	hipMalloc(&d, size);
	
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	sum << <1, count >> >(d);

	int hSum;

	hipMemcpyFromSymbol(&hSum, HIP_SYMBOL(dSum), sizeof(int));

	cout << "A soma dos valores de 1 a " << count << " igual a " << hSum << endl;

	getchar();

	hipFree(d);

	return 0;
}