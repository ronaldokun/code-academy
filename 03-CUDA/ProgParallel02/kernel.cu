
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void addArraysGPU(int* a, int* b, int* c)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main()
{
	// Constante
	const int count = 5;
	const int size = count * sizeof(int);

	// Arrays - Memória RAM
	int ha[] = { 1, 2, 3, 4, 5 };
	int hb[] = { 100, 200, 300, 400, 500 };

	// Array para gravar o resultado - Memória RAM
	int hc[count];

	// Variáveis para execução na GPU
	int *da, *db, *dc;

	// Alocação de memória na GPU
	hipMalloc(&da, size);
	hipMalloc(&db, size);
	hipMalloc(&dc, size);

	// Cópia das variáveis a e b da Memória RAM para a Memória na GPU
	hipMemcpy(da, ha, size, hipMemcpyHostToDevice);
	hipMemcpy(db, hb, size, hipMemcpyHostToDevice);

	// Definindo um bloco de threads
	addArraysGPU <<<1, count >>>(da, db, dc);

	// Cópia do resultado da Memória da GPU de volta para a Memória da CPU
	hipMemcpy(hc, dc, size, hipMemcpyDeviceToHost);

	// Imprime os resultados
	printf("%d %d %d %d %d",
		hc[0],
		hc[1],
		hc[2],
		hc[3],
		hc[4]);

	// Libera as áreas de memória
	hipFree(da);
	hipFree(db);
	hipFree(dc);

	// Para visualizar o resultado na tela até pressionar uma tecla
	getchar();

}