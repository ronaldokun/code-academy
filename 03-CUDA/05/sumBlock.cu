#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <numeric>
using namespace std;

__global__ void sumSingleBlock(int* d)
{
	int tid = threadIdx.x;

	// Número de threads participando em cada iteração 
	for (int tc = blockDim.x, stepSize = 1; tc > 0; tc >>= 1, stepSize <<= 1)
	{
		// Thread deve ter permissão para escrever. Thread ID deve ser menor que Thread Count
		if (tid < tc)
		{
			// Definindo como será a operação de Reduce
			// Precisamos especificar que a thread obteve o resultado e então somar com o próximo elemento do array
			int pa = tid * stepSize * 2;

			// Obtemos o que foi escrito (gravado) pela thread e somamos com o próximo elemento do array
			int pb = pa + stepSize;
			d[pa] += d[pb];
		}
	}
}


int main()
{
	// Status de erro
	hipError_t status;

	// Constantes
	const int count = 256;
	const int size = count * sizeof(int);

	// Definindo um array de valores inteiros 
	// Array no host
	int* h = new int[count];

	// Preenchendo o array com elementos
	for (int i = 0; i < count; ++i)
		h[i] = i + 1;

	// Array no device
	int* d;

	// Alocando device memory
	status = hipMalloc(&d, size);

	// Copiando da memória RAM para a memória do device
	status = hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	// Um bloco de thread e o máximo de threads possível em nosso caso, count/2
	sumSingleBlock << <1, count / 2, size >> >(d);

	int result;

	// Devolvendo os elementos do device para a memória RAM
	status = hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);

	cout << "Soma dos Elementos do array igual a " << result << endl;

	getchar();

	hipFree(d);
	delete[] h;

	return 0;
}

