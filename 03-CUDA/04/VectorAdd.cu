#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define AxCheckError(err) CheckError(err,__FUNCTION__, __LINE__)
#define AxCheckErrorMsg(err, msg) CheckErrorMsg(err, msg, __FUNCTION__, __LINE__)

// Gera dados de teste
void GenerateTestData(int const N, float* const a, float* const b, float* const c, float* const ref);

// Compara dados
void CompareData(int const N, float const* const a, float const* const b);

// Checa erros
void CheckError(hipError_t const err, char const* const fun, const int line);
void CheckErrorMsg(hipError_t const err, char const* const msg, char const* const fun, int const line);

// kernel
__global__ void SumArrays(float* const a, float* const b, float* const c, int const N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N)
        c[i] = a[i] + b[i];
}

int main()
{
	// Variáveis do host
    float *aH, *bH, *cH, *refH;

	// Variáveis do device
    float *aD, *bD, *cD;

	// CUDA error
    hipError_t e = hipSuccess;

	// Dimensões
    dim3 gridSize;
    dim3 blockSize;

	// Constantes
    int const N = 2053;
    int const N_BYTES = N * sizeof(float);
    int const BLOCK_SIZE = 512; 

	// Alocação de memória no host
    aH = (float*)malloc(N_BYTES);
    bH = (float*)malloc(N_BYTES);
    cH = (float*)malloc(N_BYTES);
    refH = (float*)malloc(N_BYTES);

	// Gera dados de teste
    GenerateTestData(N, aH, bH, cH, refH);

	// Aloca memória na GPU
    e = hipMalloc((void**)&aD, N_BYTES);
    AxCheckError(e);
    e = hipMalloc((void**)&bD, N_BYTES);
    AxCheckError(e);
    e = hipMalloc((void**)&cD, N_BYTES);
    AxCheckError(e);

	// Copia os valores das variáveis na memória do host para a memória do device
    e = hipMemcpy(aD, aH, N_BYTES, hipMemcpyHostToDevice);
    AxCheckError(e);
    e = hipMemcpy(bD, bH, N_BYTES, hipMemcpyHostToDevice);
    AxCheckError(e);

	// Define as dimensões
    blockSize.x = BLOCK_SIZE; blockSize.y = 1; blockSize.z = 1;
    gridSize.x = ((N + BLOCK_SIZE - 1) / BLOCK_SIZE); gridSize.y = 1; gridSize.z = 1;

	// Executa o kernel - Soma os arrays
    SumArrays<<<gridSize, blockSize>>>(aD, bD, cD, N);

    // Obtém erros de execução do kernel
    hipDeviceSynchronize();
    e = hipGetLastError();
    AxCheckError(e);

	// Copia o resultado da memória do device para a memória do host
    e = hipMemcpy(cH, cD, N_BYTES, hipMemcpyDeviceToHost);
    AxCheckError(e);

	// Compara os dados
    CompareData(N, cH, refH);

	// Libera a memória
    hipFree(aD); hipFree(bD); hipFree(cD);
    free(aH); free(bH); free(cH); free(refH);
    AxCheckError(hipDeviceReset());

	getchar();
    return 0;
}

// Função para gerar os dados
void GenerateTestData(int const N, float* const a, float* const b, float* const c, float* const ref)
{
    int i;
	
    srand((unsigned)time(NULL));

    for(i = 0; i < N; i++)
    {
        a[i] = (float)rand();
        b[i] = (float)rand();
        c[i] = 0.0f;
        ref[i] = a[i] + b[i];
    }

}

// Função para comparar os dados
void CompareData(int const N, float const* const a, float const* const b)
{
    int i;
    int different = 0;

    for(i = 0; i < N; i++)
    {
        different = (a[i] != b[i]);
        if(different)
            break;
    }

    if(different)
    {
        printf("Arrays diferentes.\n");
    }
    else
    {
        printf("Arrays match.\n");
    }

}

// Funções para checar e imprimir o código de erro
void CheckError(hipError_t const err, char const* const fun, const int line)
{
    if (err)
    {
        printf("CUDA Error Code[%d]: %s %s():%d\n",err,hipGetErrorString(err),fun,line);
        exit(1);
    }
}

void CheckErrorMsg(hipError_t const err, char const* const msg, char const* const fun, int const line)
{
    if (err)
    {
        printf("CUDA Error Code[%d]: %s %s() %d\n%s\n",err,hipGetErrorString(err),fun,line,msg);
        exit(1);
    }
}
